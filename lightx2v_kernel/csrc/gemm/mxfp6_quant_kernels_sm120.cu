#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>
#include <cuda_fp6.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <torch/all.h>

#include "utils.h"

// Get type2 from type or vice versa (applied to half and bfloat16)
template <typename T>
struct TypeConverter {
  using Type = half2;
};  // keep for generality

template <>
struct TypeConverter<half2> {
  using Type = half;
};

template <>
struct TypeConverter<half> {
  using Type = half2;
};

template <>
struct TypeConverter<__hip_bfloat162> {
  using Type = __hip_bfloat16;
};

template <>
struct TypeConverter<__hip_bfloat16> {
  using Type = __hip_bfloat162;
};

#define ELTS_PER_THREAD 8

constexpr int CVT_FP6_ELTS_PER_THREAD = 8;
constexpr int CVT_FP6_SF_VEC_SIZE = 32;

struct uint8x6_t {
  uint8_t elts[6];
};

// Convert 4 float2 values into 8 e3m2 values (represented as one uint8x6_t).
inline __device__ uint8x6_t fp32_vec_to_e3m2(float2 (&array)[4]) {
  uint64_t val;
  asm volatile(
      "{\n"
      ".reg .b16 pack0;\n"
      ".reg .b16 pack1;\n"
      ".reg .b16 pack2;\n"
      ".reg .b16 pack3;\n"
      "cvt.rn.satfinite.e3m2x2.f32   pack0, %2, %1;\n"
      "cvt.rn.satfinite.e3m2x2.f32   pack1, %4, %3;\n"
      "cvt.rn.satfinite.e3m2x2.f32   pack2, %6, %5;\n"
      "cvt.rn.satfinite.e3m2x2.f32   pack3, %8, %7;\n"
      "mov.b64 %0, {pack0, pack1, pack2, pack3};\n"
      "}"
      : "=l"(val)
      : "f"(array[0].x),
        "f"(array[0].y),
        "f"(array[1].x),
        "f"(array[1].y),
        "f"(array[2].x),
        "f"(array[2].y),
        "f"(array[3].x),
        "f"(array[3].y));

  uint8x6_t result;

  // pack 8 uint8_t into 6 uint8_t
  // here is how to pack:
  // 4个fp6 a b c d. a:[a5 a4 a3 a2 a1 a0], b..., c..., d...
  // 3个unint8 pack0 pack1 pack2
  // packed0: [b1 b0][a5 a4 a3 a2 a1 a0]
  // packed1: [c3 c2 c1 c0][b5 b4 b3 b2]
  // packed2: [d5 d4 d3 d2 d1 d0][c5 c4]

  // lower 4 uint8_t
  uint8_t l_val_0 = val & 0xFF;
  uint8_t l_val_1 = (val >> 8) & 0xFF;
  uint8_t l_val_2 = (val >> 16) & 0xFF;
  uint8_t l_val_3 = (val >> 24) & 0xFF;
  // higher 4 uint8_t
  uint8_t h_val_0 = (val >> 32) & 0xFF;
  uint8_t h_val_1 = (val >> 40) & 0xFF;
  uint8_t h_val_2 = (val >> 48) & 0xFF;
  uint8_t h_val_3 = (val >> 56) & 0xFF;

  // pack result
  result.elts[0] = (l_val_1 << 6) | l_val_0;
  result.elts[1] = (l_val_2 << 4) | (l_val_1 >> 2);
  result.elts[2] = (l_val_3 << 2) | (l_val_2 >> 4);
  result.elts[3] = (h_val_1 << 6) | h_val_0;
  result.elts[4] = (h_val_2 << 4) | (h_val_1 >> 2);
  result.elts[5] = (h_val_3 << 2) | (h_val_2 >> 4);

  return result;
}

// Fast reciprocal.
inline __device__ float reciprocal_approximate_ftz(float a) {
  float b;
  asm volatile("rcp.approx.ftz.f32 %0, %1;\n" : "=f"(b) : "f"(a));
  return b;
}

template <class SFType, int CVT_FP6_NUM_THREADS_PER_SF>
__device__ uint8_t* get_sf_out_address(int rowIdx, int colIdx, int numCols, SFType* SFout) {
// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  static_assert(CVT_FP6_NUM_THREADS_PER_SF == 4);

  // one of 4 threads write one SF to global memory.
  // TODO: stage through smem for packed STG.32
  // is it better than STG.8 from 4 threads ?
  if (threadIdx.x % CVT_FP6_NUM_THREADS_PER_SF == 0) {
    // SF vector index (32 elements share one SF in the K dimension).
    int32_t kIdx = colIdx / CVT_FP6_NUM_THREADS_PER_SF;
    int32_t mIdx = rowIdx;

    // SF layout [numMTiles, numKTiles, 32 (mTile), 4 (mTile), 4(kTile)]
    // --> index [mTileIdx, kTileIdx, outerMIdx, innerMIdx, innerKIdx]

    int32_t mTileIdx = mIdx / (32 * 4);
    // SF vector size 32.
    int factor = CVT_FP6_SF_VEC_SIZE * 4;
    int32_t numKTiles = (numCols + factor - 1) / factor;
    int64_t mTileStride = numKTiles * 32 * 4 * 4;

    int32_t kTileIdx = (kIdx / 4);
    int64_t kTileStride = 32 * 4 * 4;

    // M tile layout [32, 4] is column-major.
    int32_t outerMIdx = (mIdx % 32);    // same as (mIdx % 128) % 32
    int64_t outerMStride = 4 * 4;

    int32_t innerMIdx = (mIdx % (32 * 4)) / 32;
    int64_t innerMStride = 4;

    int32_t innerKIdx = (kIdx % 4);
    int64_t innerKStride = 1;

    // Compute the global offset.
    int64_t SFOffset = mTileIdx * mTileStride + kTileIdx * kTileStride + outerMIdx * outerMStride +
                       innerMIdx * innerMStride + innerKIdx * innerKStride;

    return reinterpret_cast<uint8_t*>(SFout) + SFOffset;
  } else {
    // Other threads do not write to SFout.
    return nullptr;
  }
}

// Define a 16 bytes packed data type.
template <class Type>
struct PackedVec {
  typename TypeConverter<Type>::Type elts[4];
};

// template <>
// struct PackedVec<__hip_fp8_e4m3_fnuz> {
//   __hip_fp8x2_e4m3_fnuz elts[8];
// };

template <class Type> // Type can be half or bfloat16
__device__ uint8x6_t cvt_warp_fp16_to_fp6(PackedVec<Type>& vec, uint8_t* SFout) {
// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  // Get absolute maximum values among the local 8 values.
  auto localMax = __habs2(vec.elts[0]);

// Local maximum value.
#pragma unroll
  for (int i = 1; i < CVT_FP6_ELTS_PER_THREAD / 2; i++) {
    localMax = __hmax2(localMax, __habs2(vec.elts[i]));
  }

  // Get the absolute maximum among all 32 values (four threads).
  localMax = __hmax2(__shfl_xor_sync(uint32_t(-1), localMax, 1), localMax);
  localMax = __hmax2(__shfl_xor_sync(uint32_t(-1), localMax, 2), localMax);
  // Get the final absolute maximum values.
  float vecMax = float(__hmax(localMax.x, localMax.y));

  // Get the SF (max value of the vector / max value of e3m2).
  // maximum value of e3m2 = 28.0.
  // TODO: use half as compute data type.
  float SFValue = (vecMax / 28.0f);
  // 8 bits representation of the SF.
  uint8_t fp8SFVal;
  // Write the SF to global memory (STG.8).
  __nv_fp8_e8m0 tmp;
  tmp.__x = __nv_cvt_float_to_e8m0(SFValue, __HIP_SATFINITE, hipRoundPosInf);
  SFValue = static_cast<float>(tmp);
  fp8SFVal = tmp.__x;


  float outputScale =
      SFValue != 0 ? reciprocal_approximate_ftz(SFValue) : 0.0f;

  if (SFout) {
    // Write the SF to global memory (STG.8).
    *SFout = fp8SFVal;
  }

  // Convert the input to float.
  float2 fp2Vals[CVT_FP6_ELTS_PER_THREAD / 2];

#pragma unroll
  for (int i = 0; i < CVT_FP6_ELTS_PER_THREAD / 2; i++) {
    if constexpr (std::is_same_v<Type, half>) {
      fp2Vals[i] = __half22float2(vec.elts[i]);
    } else {
      fp2Vals[i] = __bfloat1622float2(vec.elts[i]);
    }
    fp2Vals[i].x *= outputScale;
    fp2Vals[i].y *= outputScale;
  }

  // Convert to e3m2 values.
  uint8x6_t e3m2Vec = fp32_vec_to_e3m2(fp2Vals);

  return e3m2Vec;
}


template <class Type> // Type can be half or bfloat16
__global__ void
// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
__launch_bounds__(256, 6) cvt_fp16_to_fp6(
// #else
// cvt_fp16_to_fp6(
// #endif
    int32_t numRows, int32_t numCols, Type const* in, uint8x6_t* out, uint32_t* SFout) {
// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  using PackedVec = PackedVec<Type>;
  static constexpr int CVT_FP6_NUM_THREADS_PER_SF = (CVT_FP6_SF_VEC_SIZE / CVT_FP6_ELTS_PER_THREAD);
  static_assert(sizeof(PackedVec) == sizeof(Type) * CVT_FP6_ELTS_PER_THREAD, "Vec size is not matched.");

  // Input tensor row/col loops.
  for (int rowIdx = blockIdx.x; rowIdx < numRows; rowIdx += gridDim.x) {
    for (int colIdx = threadIdx.x; colIdx < numCols / CVT_FP6_ELTS_PER_THREAD; colIdx += blockDim.x) {
      int64_t inOffset = rowIdx * (numCols / CVT_FP6_ELTS_PER_THREAD) + colIdx;
      PackedVec in_vec = reinterpret_cast<PackedVec const*>(in)[inOffset];
      // Get the output tensor offset.
      // Same as inOffset because 8 elements(E3M2) are packed into one uint8x6_t.
      int64_t outOffset = inOffset;
      auto& out_pos = out[outOffset];

      auto sf_out =
          get_sf_out_address<uint32_t, CVT_FP6_NUM_THREADS_PER_SF>(rowIdx, colIdx, numCols, SFout);

      out_pos = cvt_warp_fp16_to_fp6<Type>(in_vec, sf_out);
    }
  }
// #endif
}

template <typename T>
void invokeFP6Quantization(
    int m,
    int n,
    T const* input,
    int64_t* output,
    int32_t* SFOuput,
    int multiProcessorCount,
    hipStream_t stream) {
  // Grid, Block size.
  // Each thread converts 8 values.
  dim3 block(std::min(int(n / ELTS_PER_THREAD), 256));
  // Get number of blocks per SM (assume we can fully utilize the SM).
  int const numBlocksPerSM = 1536 / block.x;
  dim3 grid(std::min(int(m), multiProcessorCount * numBlocksPerSM));

  // Launch the cvt kernel.
    cvt_fp16_to_fp6<T>
    <<<grid, block, 0, stream>>>(
        m, n, input, reinterpret_cast<uint8x6_t*>(output), reinterpret_cast<uint32_t*>(SFOuput));
}

// Instantiate the function.
template void invokeFP6Quantization(
    int m,
    int n,
    half const* input,
    int64_t* output,
    int32_t* SFOuput,
    int multiProcessorCount,
    hipStream_t stream);

template void invokeFP6Quantization(
    int m,
    int n,
    __hip_bfloat16 const* input,
    int64_t* output,
    int32_t* SFOuput,
    int multiProcessorCount,
    hipStream_t stream);

inline int getMultiProcessorCount() {
  static int multi_processor_count = []() {
    int device_id = 0;
    int count = 0;

    // Get the current CUDA device ID
    CHECK_CUDA_SUCCESS(hipGetDevice(&device_id));

    // Get the number of multiprocessors for the current device
    CHECK_CUDA_SUCCESS(hipDeviceGetAttribute(&count, hipDeviceAttributeMultiprocessorCount, device_id));

    return count;  // Initialize the static variable
  }();

  return multi_processor_count;  // Return the cached value on subsequent calls
}

void scaled_fp6_quant_sm120(
    torch::Tensor& output, torch::Tensor const& input, torch::Tensor& output_sf) {
  int32_t m = input.size(0);
  int32_t n = input.size(1);

  TORCH_CHECK(n % 32 == 0, "The N dimension must be multiple of 32.");

  int multiProcessorCount = getMultiProcessorCount();

  auto sf_out = static_cast<int32_t*>(output_sf.data_ptr());
  auto output_ptr = static_cast<int64_t*>(output.data_ptr());
  at::cuda::CUDAGuard device_guard{(char)input.get_device()};
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(input.get_device());

  switch (input.scalar_type()) {
    case torch::kHalf: {
      auto input_ptr = reinterpret_cast<half const*>(input.data_ptr());
      invokeFP6Quantization(m, n, input_ptr, output_ptr, sf_out, multiProcessorCount, stream);
      break;
    }
    case torch::kBFloat16: {
      auto input_ptr = reinterpret_cast<__hip_bfloat16 const*>(input.data_ptr());
      invokeFP6Quantization(m, n, input_ptr, output_ptr, sf_out, multiProcessorCount, stream);
      break;
    }
    default: {
      std::cerr << "Observing: " << input.scalar_type() << " for the input datatype which is invalid";
      throw std::runtime_error("Unsupported input data type for quantize_to_fp6.");
    }
  }
}
