#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <torch/all.h>

#include "utils.h"

// Get type2 from type or vice versa (applied to half and bfloat16)
template <typename T>
struct TypeConverter {
  using Type = half2;
};  // keep for generality

template <>
struct TypeConverter<half2> {
  using Type = half;
};

template <>
struct TypeConverter<half> {
  using Type = half2;
};

template <>
struct TypeConverter<__hip_bfloat162> {
  using Type = __hip_bfloat16;
};

template <>
struct TypeConverter<__hip_bfloat16> {
  using Type = __hip_bfloat162;
};

#define ELTS_PER_THREAD 8

constexpr int CVT_FP8_ELTS_PER_THREAD = 8;
constexpr int CVT_FP8_SF_VEC_SIZE = 32;


// Convert 4 float2 values into 8 e4m3 values (represented as one uint64_t).
inline __device__ uint64_t fp32_vec_to_e4m3(float2 (&array)[4]) {
  uint64_t val;
  asm volatile(
      "{\n"
      ".reg .b16 pack0;\n"
      ".reg .b16 pack1;\n"
      ".reg .b16 pack2;\n"
      ".reg .b16 pack3;\n"
      "cvt.rn.satfinite.e4m3x2.f32   pack0, %2, %1;\n"
      "cvt.rn.satfinite.e4m3x2.f32   pack1, %4, %3;\n"
      "cvt.rn.satfinite.e4m3x2.f32   pack2, %6, %5;\n"
      "cvt.rn.satfinite.e4m3x2.f32   pack3, %8, %7;\n"
      "mov.b64 %0, {pack0, pack1, pack2, pack3};\n"
      "}"
      : "=l"(val)
      : "f"(array[0].x),
        "f"(array[0].y),
        "f"(array[1].x),
        "f"(array[1].y),
        "f"(array[2].x),
        "f"(array[2].y),
        "f"(array[3].x),
        "f"(array[3].y));
  return val;
}

// Fast reciprocal.
inline __device__ float reciprocal_approximate_ftz(float a) {
  float b;
  asm volatile("rcp.approx.ftz.f32 %0, %1;\n" : "=f"(b) : "f"(a));
  return b;
}

template <class SFType, int CVT_FP8_NUM_THREADS_PER_SF>
__device__ uint8_t* get_sf_out_address(int rowIdx, int colIdx, int numCols, SFType* SFout) {
// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  static_assert(CVT_FP8_NUM_THREADS_PER_SF == 4);

  // one of 4 threads write one SF to global memory.
  // TODO: stage through smem for packed STG.32
  // is it better than STG.8 from 4 threads ?
  if (threadIdx.x % CVT_FP8_NUM_THREADS_PER_SF == 0) {
    // SF vector index (16 elements share one SF in the K dimension).
    int32_t kIdx = colIdx / CVT_FP8_NUM_THREADS_PER_SF;
    int32_t mIdx = rowIdx;

    // SF layout [numMTiles, numKTiles, 32 (mTile), 4 (mTile), 4(kTile)]
    // --> index [mTileIdx, kTileIdx, outerMIdx, innerMIdx, innerKIdx]

    int32_t mTileIdx = mIdx / (32 * 4);
    // SF vector size 32.
    int factor = CVT_FP8_SF_VEC_SIZE * 4;
    int32_t numKTiles = (numCols + factor - 1) / factor;
    int64_t mTileStride = numKTiles * 32 * 4 * 4;

    int32_t kTileIdx = (kIdx / 4);
    int64_t kTileStride = 32 * 4 * 4;

    // M tile layout [32, 4] is column-major.
    int32_t outerMIdx = (mIdx % 32);    // same as (mIdx % 128) % 32
    int64_t outerMStride = 4 * 4;

    int32_t innerMIdx = (mIdx % (32 * 4)) / 32;
    int64_t innerMStride = 4;

    int32_t innerKIdx = (kIdx % 4);
    int64_t innerKStride = 1;

    // Compute the global offset.
    int64_t SFOffset = mTileIdx * mTileStride + kTileIdx * kTileStride + outerMIdx * outerMStride +
                       innerMIdx * innerMStride + innerKIdx * innerKStride;

    return reinterpret_cast<uint8_t*>(SFout) + SFOffset;
  } else {
    // Other threads do not write to SFout.
    return nullptr;
  }
}

// Define a 16 bytes packed data type.
template <class Type>
struct PackedVec {
  typename TypeConverter<Type>::Type elts[4];
};

template <>
struct PackedVec<__hip_fp8_e4m3_fnuz> {
  __hip_fp8x2_e4m3_fnuz elts[8];
};

// Quantizes the provided PackedVec into the uint64_t output
template <class Type> // Type can be half or bfloat16
__device__ uint64_t cvt_warp_fp16_to_fp8(PackedVec<Type>& vec, uint8_t* SFout) {
// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  // Get absolute maximum values among the local 8 values.
  auto localMax = __habs2(vec.elts[0]);

// Local maximum value.
#pragma unroll
  for (int i = 1; i < CVT_FP8_ELTS_PER_THREAD / 2; i++) {
    localMax = __hmax2(localMax, __habs2(vec.elts[i]));
  }

  // Get the absolute maximum among all 32 values (four threads).
  localMax = __hmax2(__shfl_xor_sync(uint32_t(-1), localMax, 1), localMax);
  localMax = __hmax2(__shfl_xor_sync(uint32_t(-1), localMax, 2), localMax);
  // Get the final absolute maximum values.
  float vecMax = float(__hmax(localMax.x, localMax.y));

  // Get the SF (max value of the vector / max value of e4m3).
  // maximum value of e4m3 = 448.0.
  // TODO: use half as compute data type.
  float SFValue = (vecMax / 448.0f);
  // 8 bits representation of the SF.
  uint8_t fp8SFVal;
  // Write the SF to global memory (STG.8).
  __nv_fp8_e8m0 tmp;
  tmp.__x = __nv_cvt_float_to_e8m0(SFValue, __HIP_SATFINITE, hipRoundPosInf);
  SFValue = static_cast<float>(tmp);
  fp8SFVal = tmp.__x;


  float outputScale =
      SFValue != 0 ? reciprocal_approximate_ftz(SFValue) : 0.0f;

  if (SFout) {
    // Write the SF to global memory (STG.8).
    *SFout = fp8SFVal;
  }

  // Convert the input to float.
  float2 fp2Vals[CVT_FP8_ELTS_PER_THREAD / 2];

#pragma unroll
  for (int i = 0; i < CVT_FP8_ELTS_PER_THREAD / 2; i++) {
    if constexpr (std::is_same_v<Type, half>) {
      fp2Vals[i] = __half22float2(vec.elts[i]);
    } else {
      fp2Vals[i] = __bfloat1622float2(vec.elts[i]);
    }
    fp2Vals[i].x *= outputScale;
    fp2Vals[i].y *= outputScale;
  }

  // Convert to e4m3 values.
  uint64_t e4m3Vec = fp32_vec_to_e4m3(fp2Vals);

  return e4m3Vec;
}


template <class Type> // Type can be half or bfloat16
__global__ void
// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
__launch_bounds__(256, 6) cvt_fp16_to_fp8(
// #else
// cvt_fp16_to_fp8(
// #endif
    int32_t numRows, int32_t numCols, Type const* in, uint64_t* out, uint32_t* SFout) {
// #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  using PackedVec = PackedVec<Type>;
  static constexpr int CVT_FP8_NUM_THREADS_PER_SF = (CVT_FP8_SF_VEC_SIZE / CVT_FP8_ELTS_PER_THREAD);
  static_assert(sizeof(PackedVec) == sizeof(Type) * CVT_FP8_ELTS_PER_THREAD, "Vec size is not matched.");

  // Input tensor row/col loops.
  for (int rowIdx = blockIdx.x; rowIdx < numRows; rowIdx += gridDim.x) {
    for (int colIdx = threadIdx.x; colIdx < numCols / CVT_FP8_ELTS_PER_THREAD; colIdx += blockDim.x) {
      int64_t inOffset = rowIdx * (numCols / CVT_FP8_ELTS_PER_THREAD) + colIdx;
      PackedVec in_vec = reinterpret_cast<PackedVec const*>(in)[inOffset];
      // Get the output tensor offset.
      // Same as inOffset because 8 elements(E4M3) are packed into one uint64_t.
      int64_t outOffset = inOffset;
      auto& out_pos = out[outOffset];

      auto sf_out =
          get_sf_out_address<uint32_t, CVT_FP8_NUM_THREADS_PER_SF>(rowIdx, colIdx, numCols, SFout);

      out_pos = cvt_warp_fp16_to_fp8<Type>(in_vec, sf_out);
    }
  }
// #endif
}

template <typename T>
void invokeFP8Quantization(
    int m,
    int n,
    T const* input,
    int64_t* output,
    int32_t* SFOuput,
    int multiProcessorCount,
    hipStream_t stream) {
  // Grid, Block size.
  // Each thread converts 8 values.
  dim3 block(std::min(int(n / ELTS_PER_THREAD), 256));
  // Get number of blocks per SM (assume we can fully utilize the SM).
  int const numBlocksPerSM = 1536 / block.x;
  dim3 grid(std::min(int(m), multiProcessorCount * numBlocksPerSM));

  // Launch the cvt kernel.
    cvt_fp16_to_fp8<T>
    <<<grid, block, 0, stream>>>(
        m, n, input, reinterpret_cast<uint64_t*>(output), reinterpret_cast<uint32_t*>(SFOuput));
}

// Instantiate the function.
template void invokeFP8Quantization(
    int m,
    int n,
    half const* input,
    int64_t* output,
    int32_t* SFOuput,
    int multiProcessorCount,
    hipStream_t stream);

template void invokeFP8Quantization(
    int m,
    int n,
    __hip_bfloat16 const* input,
    int64_t* output,
    int32_t* SFOuput,
    int multiProcessorCount,
    hipStream_t stream);

inline int getMultiProcessorCount() {
  static int multi_processor_count = []() {
    int device_id = 0;
    int count = 0;

    // Get the current CUDA device ID
    CHECK_CUDA_SUCCESS(hipGetDevice(&device_id));

    // Get the number of multiprocessors for the current device
    CHECK_CUDA_SUCCESS(hipDeviceGetAttribute(&count, hipDeviceAttributeMultiprocessorCount, device_id));

    return count;  // Initialize the static variable
  }();

  return multi_processor_count;  // Return the cached value on subsequent calls
}

void scaled_fp8_quant_sm120(
    torch::Tensor& output, torch::Tensor const& input, torch::Tensor& output_sf) {
  int32_t m = input.size(0);
  int32_t n = input.size(1);

  TORCH_CHECK(n % 32 == 0, "The N dimension must be multiple of 16.");

  int multiProcessorCount = getMultiProcessorCount();

  auto sf_out = static_cast<int32_t*>(output_sf.data_ptr());
  auto output_ptr = static_cast<int64_t*>(output.data_ptr());
  at::cuda::CUDAGuard device_guard{(char)input.get_device()};
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(input.get_device());

  switch (input.scalar_type()) {
    case torch::kHalf: {
      auto input_ptr = reinterpret_cast<half const*>(input.data_ptr());
      invokeFP8Quantization(m, n, input_ptr, output_ptr, sf_out, multiProcessorCount, stream);
      break;
    }
    case torch::kBFloat16: {
      auto input_ptr = reinterpret_cast<__hip_bfloat16 const*>(input.data_ptr());
      invokeFP8Quantization(m, n, input_ptr, output_ptr, sf_out, multiProcessorCount, stream);
      break;
    }
    default: {
      std::cerr << "Observing: " << input.scalar_type() << " for the input datatype which is invalid";
      throw std::runtime_error("Unsupported input data type for quantize_to_fp8.");
    }
  }
}
